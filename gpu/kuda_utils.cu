//------------------------------------------------------------------------
// kuda_utils.cpp - GPU utility functions
//------------------------------------------------------------------------
// Author: Lam H. Dao <daohailam(at)yahoo(dot)com>
//------------------------------------------------------------------------
#include "kuda_utils.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//------------------------------------------------------------------------
void PrintCudaError(const char *fx, int n)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		printf("[%s @ %d] CUDA error: %s.", fx, n, hipGetErrorString(err));
	}
}
//------------------------------------------------------------------------
bool CudaInit(int devNo)
{
	int n, v;

	if (hipGetDeviceCount(&n) != hipSuccess) {
		printf("Get dev count error!!!!\n", 0);
		return false;
	}

	if (devNo >= n) {
		printf("Cannot set Cuda Device #%d.\n", devNo);
		return false;
	}

	hipSetDevice(devNo);
	if (hipDriverGetVersion(&v) != hipSuccess)
		return false;

	printf("* CUDA:\n  o Toolkit/Driver version: %d / %d\n", CUDA_VERSION, v);
	if ((v / 1000) < (CUDA_VERSION/1000)) {
		return false;
	}
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devNo);
	printf("  o Device name: %s.\n", prop.name);
	if (!prop.canMapHostMemory) {
		printf("Cannot use Host memory map.\n", 0);
	} else {
		hipSetDeviceFlags(hipDeviceMapHost);
	}
	#define S	"    "
	printf(S"- GPU integrated: %s.\n", prop.integrated ? "Yes" : "No");
	printf(S"- WarpSize = %d\n", prop.warpSize);
	printf(S"- SharedMemPerBlock = %d\n", prop.sharedMemPerBlock);
	printf(S"- MaxThreadsPerBlock = %d\n", prop.maxThreadsPerBlock);
	printf(S"- MaxGridSize = %d %d %d\n",
			prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf(S"- MultiProcessorCount = %d\n", prop.multiProcessorCount);
	printf(S"- MaxThreadsPerMultiProcessor = %d\n\n",
			prop.maxThreadsPerMultiProcessor);

	return n > 0;
}
//------------------------------------------------------------------------
